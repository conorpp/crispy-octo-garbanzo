#include <err.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

#define BUFLEN (4096)

int
main(int argc, char *argv[])
{
	unsigned char host_buf[BUFLEN];
	unsigned char *gpu_buf;

	while (hipMalloc(&gpu_buf, BUFLEN) == hipSuccess) {
		if (hipMemcpy(host_buf, gpu_buf, BUFLEN, hipMemcpyDeviceToHost))
			err(1, "hipMemcpy failed\n");
		write(STDOUT_FILENO, host_buf, BUFLEN);
	}

	return 0;
}
