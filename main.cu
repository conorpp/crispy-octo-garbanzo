#include <err.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

#define BUFLEN (4096)

int byte_sum(uint8_t* bytes, int size)
{
    int i = 0;
    int sum = 0;
    for(; i< size; i++)
    {
        sum += bytes[i];
    }
    return sum;
}
void print_nonzeros(uint8_t* bytes, int size)
{
    int i = 0;
    for(; i< size; i++)
    {
        if (bytes[i] != 0)
            printf("%x", bytes[i]);
    }
    return;
}
    int
main(int argc, char *argv[])
{
    unsigned char host_buf[BUFLEN];
    unsigned char *gpu_buf;
    uint64_t num_bytes =0;

    while (hipMalloc(&gpu_buf, BUFLEN) == hipSuccess) {
        num_bytes+=BUFLEN;
        if (hipMemcpy(host_buf, gpu_buf, BUFLEN, hipMemcpyDeviceToHost))
            err(1, "hipMemcpy failed\n");
    
        if (byte_sum(host_buf, BUFLEN) < 768)
        {
            continue;
        }
        write(fileno(stdout), host_buf, BUFLEN);
        printf("\n");
    }
    printf("%lld bytes copied\n",num_bytes);

    return 0;
}
